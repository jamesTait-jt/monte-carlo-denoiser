#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <SDL2/SDL.h>
#include <omp.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <chrono>

#include "constants/config.h"
#include "constants/materials.h"
#include "mcr.h"
#include "triangle.h"
#include "sphere.h"
#include "util.h"

int main (int argc, char* argv[]) {
    
    // Calculate the dimensions of the supersampled image
    const int supersample_width = screen_width * anti_aliasing_factor;
    const int supersample_height = screen_height * anti_aliasing_factor;


    // ----- IMAGE ----- //

    // Pointer to the image on the host (CPU)
    vec3 * h_output = new vec3[supersample_width * supersample_height];
    // Pointer to the image on the device (GPU)
    vec3 * d_output;

    // Pointer to the aliased image on the host (CPU)
    vec3 * h_aliased_output = new vec3[screen_width * screen_height];
    // Pointer to the aliased image on the device (GPU)
    vec3 * d_aliased_output;

    // ----- FEATURE BUFFERS ----- //

    // Pointer to the surface normals on host
    vec3 * h_surface_normals = new vec3[supersample_width * supersample_height];
    // Pointer to the surface normals on device
    vec3 * d_surface_normals;

    // Pointer to the albedo buffer on host
    vec3 * h_albedos = new vec3[supersample_width * supersample_height];
    // Pointer to the albedo buffer on the device
    vec3 * d_albedos;

    // Pointer to the depth buffer on host
    float * h_depths = new float[supersample_width * supersample_height];
    // Pointer to the depth buffer on device
    float * d_depths;

    // ----- VARIANCES ----- //

    // Pointer to the colour variances on host 
    float * h_colour_variances = new float[supersample_width * supersample_height];
    // Pointer to the colour variances on device 
    float * d_colour_variances;

    // Pointer to the surface normals on host
    float * h_surface_normal_variances = new float[supersample_width * supersample_height];
    // Pointer to the surface normals on device
    float * d_surface_normal_variances;

    // Pointer to the albedo buffer on host
    float * h_albedo_variances = new float[supersample_width * supersample_height];
    // Pointer to the albedo buffer on the device
    float * d_albedo_variances;

    // Pointer to the depth buffer on host
    float * h_depth_variances = new float[supersample_width * supersample_height];
    // Pointer to the depth buffer on device
    float * d_depth_variances;

    // Allocate memory on CUDA device
    hipMalloc(&d_output, supersample_width * supersample_height * sizeof(vec3));
    hipMalloc(&d_aliased_output, screen_width * screen_height * sizeof(vec3));

    hipMalloc(&d_surface_normals, supersample_width * supersample_height * sizeof(vec3));
    hipMalloc(&d_albedos, supersample_width * supersample_height * sizeof(vec3));
    hipMalloc(&d_depths, supersample_width * supersample_height * sizeof(float));

    hipMalloc(&d_colour_variances, supersample_width * supersample_height * sizeof(float));
    hipMalloc(&d_surface_normal_variances, supersample_width * supersample_height * sizeof(float));
    hipMalloc(&d_albedo_variances, supersample_width * supersample_height * sizeof(float));
    hipMalloc(&d_depth_variances, supersample_width * supersample_height * sizeof(float));

    // Specify the block and grid dimensions to schedule CUDA threads
    dim3 threads_per_block(8, 8);
    dim3 num_blocks(
        supersample_width / threads_per_block.x,
        supersample_height / threads_per_block.y
    );

    // Create a vector of random states for use on the device
    hiprandState * d_rand_states;
    hipMalloc(
        (void **)&d_rand_states,
        supersample_width * supersample_height * samples_per_pixel * sizeof(hiprandState)
    );

    // Load in the shapes
    int num_tris = 32;
    Triangle * triangles;
    hipMallocManaged(&triangles, num_tris * sizeof(Triangle));

    int num_spheres = 1;
    Sphere * spheres;
    hipMallocManaged(&spheres, num_tris * sizeof(Sphere));

    printf("CUDA has been initialised. Begin rendering...\n");
    printf("=============================================\n\n");

    // Load the polygons into the triangles array
    loadShapes(triangles, spheres);

    // Define our area light
    LightSphere light_sphere(
        light_start_position, 
        area_light_radius, 
        num_lights, 
        light_intensity, 
        light_colour
    );

    vec4 * camera_start_positions = new vec4[num_iterations];
    float * camera_start_yaws = new float[num_iterations];

    srand(time(NULL));
    generateCameraStartPositions(camera_start_positions, camera_start_yaws);

    //SdlWindowHelper sdl_window(screen_width, screen_height);

    for(int i = 0 ; i < num_iterations ; i++) {

        if (num_iterations == 1) {
            camera_start_positions[0] = cam_start_position;
            camera_start_yaws[0] = cam_start_yaw;
        }

        // Initialise the camera object
        Camera camera(
            camera_start_positions[i],
            camera_start_yaws[i],
            cam_focal_length
        );

        auto start = std::chrono::high_resolution_clock::now();

        // Launch the CUDA kernel from the host and begin rendering
        render_init <<<num_blocks, threads_per_block>>>(
            d_rand_states,
            supersample_height,
            supersample_width
        );

        render_kernel <<<num_blocks, threads_per_block>>>(
            d_output,
            supersample_height,
            supersample_width,
            camera,
            light_sphere,
            triangles,
            num_tris,
            spheres,
            num_spheres,
            d_rand_states
        );

        // Copy results of rendering back to the host
        hipMemcpy(
            h_output,
            d_output,
            supersample_width * supersample_height * sizeof(vec3),
            hipMemcpyDeviceToHost
        );

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = end - start;
        int duration_in_ms = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

        printf("Finished rendering in %dms.\n", duration_in_ms);

        save_image(
            h_output,
            supersample_height,
            supersample_width,
            pre_alias_title + "-" + std::to_string(i)
        );

        // Specify different scheduling, this time we assign a thread to each pixel
        // of the output image
        threads_per_block = dim3(8, 8);
        num_blocks = dim3(
            screen_width / threads_per_block.x,
            screen_height / threads_per_block.y
        );

        // Perform anti aliasing
        MSAA<<<num_blocks, threads_per_block>>>(
            d_output,
            d_aliased_output,
            supersample_height,
            supersample_width
        );

        // Copy results of rendering back to the host
        hipMemcpy(
            h_aliased_output,
            d_aliased_output,
            screen_width * screen_height * sizeof(vec3),
            hipMemcpyDeviceToHost
        );

        printf("Finished aliasing!\n");

        // Save the aliased image
        save_image(
                h_aliased_output,
                screen_height,
                screen_width,
                aliased_title + "-" + std::to_string(i)
        );

        /*
        save_patches(
            h_aliased_output,
            patch_size
        );
        */

        /*
        view_live(
            h_aliased_output,
            sdl_window
        );
        */

    }

    // Free CUDA memory
    hipFree(d_output);
    hipFree(d_aliased_output);
    hipFree(d_rand_states);

    hipFree(d_output);
    hipFree(d_aliased_output);

    hipFree(d_surface_normals);
    hipFree(d_albedos);
    hipFree(d_depths);

    hipFree(d_colour_variances);
    hipFree(d_surface_normal_variances);
    hipFree(d_albedo_variances);
    hipFree(d_depth_variances);

    hipFree(triangles);
    hipFree(spheres);

    // Clear memory for host
    delete[] h_output;
    delete[] h_aliased_output;

    delete[] h_output;
    delete[] h_aliased_output;

    delete[] h_surface_normals;
    delete[] h_albedos;
    delete[] h_depths;

    delete[] h_colour_variances;
    delete[] h_surface_normal_variances;
    delete[] h_albedo_variances;
    delete[] h_depth_variances;

    return 0;
}

void view_live(
    vec3 * image,
    SdlWindowHelper sdl_helper
) {
    for (int i = 0 ; i < screen_width * screen_height ; i++) {
        int x = i % screen_width;
        int y = i / screen_width;
        sdl_helper.putPixel(x, y, image[i]);
    }
    sdl_helper.render();
}

// Generates a list of starting positions for the camera and fills the array
void generateCameraStartPositions(
    vec4 * camera_start_positions,
    float * camera_start_yaws
) {
    for (int i = 0 ; i < num_iterations ; i++) {
        int min = -1;
        int max = 1;

        float randx = min + ((float) rand() / (float) RAND_MAX) * (max - min);
        float randy = min + ((float) rand() / (float) RAND_MAX) * (max - min);
        float randz = min + ((float) rand() / (float) RAND_MAX) * (max - min);

        min = 0;
        max = 2 * (float) M_PI;

        float rand_yaw = min + ((float) rand() / (float) RAND_MAX) * (max - min);

        camera_start_positions[i] = vec4(randx, randy, randz, 1.0f);
        camera_start_yaws[i] = rand_yaw;
    }
}

// Initialises the random states for each thread with the same seed
__global__ 
void render_init(
    hiprandState * rand_state,
    int supersample_width,
    int supersample_height
) {
    // Assign a thread to each pixel (x, y)
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the pixel index in the linearised array
    unsigned int pixel_index = (supersample_height - y - 1) * supersample_width + x;

    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1927, pixel_index, 0, &rand_state[pixel_index]);
}

// Bulk of the rendering is controlled here
__global__
void render_kernel(
    vec3 * output,
    int supersample_width,
    int supersample_height,
    Camera camera,
    LightSphere light_sphere,
    Triangle * triangles,
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    hiprandState * rand_state
) {
    // Assign a cuda thread to each pixel (x,y)
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // The index of the pixel we are working on when the 2x2 array is linearised
    unsigned int pixel_index = (supersample_height - y - 1) * supersample_width + x;


    // Flip the y coordinate
    y = supersample_height - y;

    // Change the ray's direction to work for the current pixel (pixel space -> Camera space)
    vec4 dir(
        (float)x - supersample_width / 2 , 
        (float)y - supersample_height / 2 , 
        camera.focal_length_,
        1
    ); 

    vec3 final_estimate = vec3(0.0f);
    for (int i = 0 ; i < samples_per_pixel ; i++) {
        // Get the rand state for this thread
        hiprandState local_rand_state = rand_state[pixel_index];
        // Create a ray for the given pixel
        Ray ray(camera.position_, dir);
        ray.rotateRay(camera.yaw_);
        vec3 estimate = ray.tracePath(
            triangles,
            num_tris,
            spheres,
            num_spheres,
            local_rand_state,
            monte_carlo_max_depth,
            0
        );
        printf("%d %d %f %f %f\n", pixel_index, i, estimate.x, estimate.y, estimate.z);
        final_estimate += estimate;
    }
    output[pixel_index] = final_estimate / (float) samples_per_pixel;

        // If the ray intersects with an object in the scene, perform monte carlo to
        // obtain a lighting estimate
        /*
        if (ray.closestIntersection(triangles, num_tris, spheres, num_spheres)) {

            vec3 colour = tracePath(
                    ray.closest_intersection_,
                    triangles,
                    num_tris,
                    spheres,
                    num_spheres,
                    local_rand_state,
                    monte_carlo_max_depth,
                    0
            );
            output[pixel_index] += colour;
        }
        // if there is no intersection, we set the colour to be black
        else {
            output[pixel_index] = vec3(0.0f);
        }
        */
}

// Calculates the indirect and direct light estimation for diffuse objects
__device__
vec3 monteCarlo(
    Intersection closest_intersection, 
    Triangle * triangles, 
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    LightSphere light_sphere,
    hiprandState rand_state,
    int max_depth,
    int depth
) {
    // If we have exceeded our limit of recursion, return the direct light at
    // this point multiplied by the object's colour
    if (depth >= max_depth) {
        vec3 direct_light = light_sphere.directLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres
        );

        vec3 base_colour;
        if (closest_intersection.is_triangle) {
            base_colour = triangles[closest_intersection.index].material_.diffuse_light_component_;
        } else {
            base_colour = spheres[closest_intersection.index].material_.diffuse_light_component_;
        }
        return direct_light * base_colour;
    } 
    // Otherwise, we must obtain an indirect lighting estimate for this point
    else {
        vec3 base_colour;
        if (closest_intersection.is_triangle) {
            base_colour = triangles[closest_intersection.index].material_.diffuse_light_component_;
        } else {
            base_colour = spheres[closest_intersection.index].material_.diffuse_light_component_;
        }
        vec3 direct_light = light_sphere.directLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres
        );
        vec3 indirect_estimate = indirectLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres,
            light_sphere,
            rand_state,
            max_depth,
            depth + 1
        );
        return (direct_light + indirect_estimate) * base_colour;
    }
}

__device__
vec3 indirectLight(
    Intersection closest_intersection, 
    Triangle * triangles, 
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    LightSphere light_sphere,
    hiprandState rand_state,
    int max_depth,
    int depth
) {
    vec3 intersection_normal_3 = vec3(closest_intersection.normal);
    
    vec3 N_t, N_b;
    createCoordinateSystem(intersection_normal_3, N_t, N_b);

    vec3 indirect_estimate = vec3(0);
    float pdf = 1 / (2 * (float)M_PI);
    for (int i = 0 ; i < monte_carlo_num_samples ; i++) {
        float r1 = hiprand_uniform(&rand_state); // cos(theta) = N.Light Direction
        float r2 = hiprand_uniform(&rand_state);
        vec3 sample = uniformSampleHemisphere(r1, r2);

        // Convert the sample from our coordinate space to world space
        vec4 sample_world(
            sample.x * N_b.x + sample.y * intersection_normal_3.x + sample.z * N_t.x,
            sample.x * N_b.y + sample.y * intersection_normal_3.y + sample.z * N_t.y,
            sample.x * N_b.z + sample.y * intersection_normal_3.z + sample.z * N_t.z,
            0
        );

        // Generate our ray from the random direction calculated previously
        Ray random_ray(
            closest_intersection.position + sample_world * 0.0001f,
            sample_world
        );

        if (random_ray.closestIntersection(triangles, num_tris, spheres, num_spheres)) {
               indirect_estimate += r1 * monteCarlo(
               random_ray.closest_intersection_,
               triangles,
               num_tris,
               spheres,
               num_spheres,
               light_sphere,
               rand_state,
               max_depth,
               depth + 1
            );
        }
    } 
    indirect_estimate /= monte_carlo_num_samples * pdf;
    return indirect_estimate;
}



__global__
void MSAA(
    vec3 * supersampled_image,
    vec3 * aliased_output,
    int supersample_height,
    int supersample_width
) {
    // Assign a cuda thread to each pixel (x,y)
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // The index of the pixel we are working on when the 2x2 array is linearised
    unsigned int output_pixel_index = (screen_height - y - 1) * screen_width + x;

    // Multiply x and y by the factor so that every pixel is included
    y *= anti_aliasing_factor;
    x *= anti_aliasing_factor;

    // Average the pixel values in a (factor^2 by factor^2) grid
    vec3 avg_pixel_value(0.0f);
    for (int i = 0 ; i < anti_aliasing_factor ; i++) {
        for (int j = 0 ; j < anti_aliasing_factor ; j++) {
            unsigned int input_pixel_index = (supersample_height - (y + i) - 1) * supersample_width + (x + j);
            avg_pixel_value += supersampled_image[input_pixel_index] / (float)(anti_aliasing_factor * anti_aliasing_factor);
        }
    }
    aliased_output[output_pixel_index] = avg_pixel_value;
}

void update(Camera & camera, Light & light) {
    static int t = SDL_GetTicks();
    /* Compute frame time */
    int t2 = SDL_GetTicks();
    float dt = float(t2-t);
    t = t2;

    std::cout << "Render time: " << dt << "ms." << std::endl;

    /* Update variables*/

    const Uint8* keystate = SDL_GetKeyboardState(NULL);

    if (keystate[SDL_SCANCODE_UP]) {
        camera.moveForwards(0.1);
    }
    if (keystate[SDL_SCANCODE_DOWN]) {
        camera.moveBackwards(0.1);
    }
    if (keystate[SDL_SCANCODE_LEFT]) {
        camera.rotateLeft(0.1);
    }
    if (keystate[SDL_SCANCODE_RIGHT]) {
        camera.rotateRight(0.1);
    }
    /*if (keystate[SDL_SCANCODE_A]) {
    }
    if (keystate[SDL_SCANCODE_D]) {
        light.translateRight(0.1);
    }
    if (keystate[SDL_SCANCODE_Q]) {
        light.translateUp(0.1);
    }
    if (keystate[SDL_SCANCODE_E]) {
        light.translateDown(0.1);
    }
    if (keystate[SDL_SCANCODE_W]) {
        light.translateForwards(0.1);
    }
    if (keystate[SDL_SCANCODE_S]) {
        light.translateBackwards(0.1);
    }*/
}

void loadShapes(Triangle * triangles, Sphere * spheres) {
    float cornell_length = 555;			// Length of Cornell Box side.

    vec4 A(cornell_length, 0, 0             , 1);
    vec4 B(0             , 0, 0             , 1);
    vec4 C(cornell_length, 0, cornell_length, 1);
    vec4 D(0             , 0, cornell_length, 1);

    vec4 E(cornell_length, cornell_length, 0             , 1);
    vec4 F(0             , cornell_length, 0             , 1);
    vec4 G(cornell_length, cornell_length, cornell_length, 1);
    vec4 H(0             , cornell_length, cornell_length, 1);

    // Counter to track triangles
    int curr_tris = 0;

    // Triangles now take a material as an argument rather than a colour
    // Floor:
    Triangle floor_triangle_1 = Triangle(C, B, A, m_sol_base3);
    //triangles.push_back(floor_triangle_1);
    triangles[curr_tris] = floor_triangle_1;
    curr_tris++;

    Triangle floor_triangle_2 = Triangle(C, D, B, m_sol_base3);
    //triangles.push_back(floor_triangle_2);
    triangles[curr_tris] = floor_triangle_2;
    curr_tris++;

    // Left wall
    Triangle left_wall_1 = Triangle(A, E, C, m_sol_base02);
    //triangles.push_back(left_wall_1);
    triangles[curr_tris] = left_wall_1;
    curr_tris++;

    Triangle left_wall_2 = Triangle(C, E, G, m_sol_base02);
    //triangles.push_back(left_wall_2);
    triangles[curr_tris] = left_wall_2;
    curr_tris++;

    // Right wall
    Triangle right_wall_1 = Triangle(F, B, D, m_sol_base02);
    //triangles.push_back(right_wall_1);
    triangles[curr_tris] = right_wall_1;
    curr_tris++;

    Triangle right_wall_2 = Triangle(H, F, D, m_sol_base02);
    //triangles.push_back(right_wall_2);
    triangles[curr_tris] = right_wall_2;
    curr_tris++;

    // Ceiling
    Triangle ceiling_1 = Triangle(E, F, G, m_sol_base01);
    //triangles.push_back(ceiling_1);
    triangles[curr_tris] = ceiling_1;
    curr_tris++;

    Triangle ceiling_2 = Triangle(F, H, G, m_sol_base01);
    //triangles.push_back(ceiling_2);
    triangles[curr_tris] = ceiling_2;
    curr_tris++;

    // Back wall
    Triangle back_wall_1 = Triangle(G, D, C, m_sol_yellow);
    //triangles.push_back(back_wall_1);
    triangles[curr_tris] = back_wall_1;
    curr_tris++;

    Triangle back_wall_2 = Triangle(G, H, D, m_sol_yellow);
    //triangles.push_back(back_wall_2);
    triangles[curr_tris] = back_wall_2;
    curr_tris++;

    if (num_iterations != 1) {
        // Front Wall
        Triangle front_wall_1 = Triangle(A, E, F, m_sol_orange);
        triangles[curr_tris] = front_wall_1;
        curr_tris++;

        Triangle front_wall_2 = Triangle(A, F, B, m_sol_orange);
        triangles[curr_tris] = front_wall_2;
        curr_tris++;
    }


    // ---------------------------------------------------------------------------
    // Short block

    A = vec4(240,0,234,1);  //+120 in z -50 in x
    B = vec4( 80,0,185,1);
    C = vec4(190,0,392,1);
    D = vec4( 32,0,345,1);

    E = vec4(240,165,234,1);
    F = vec4( 80,165,185,1);
    G = vec4(190,165,392,1);
    H = vec4( 32,165,345,1);

    // Front
    //triangles.push_back(Triangle(E, B, A, m_sol_red));
    triangles[curr_tris] = Triangle(E, B, A, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(E, F, B, m_sol_red));
    triangles[curr_tris] = Triangle(E, F, B, m_sol_red);
    curr_tris++;

    // Front
    //triangles.push_back(Triangle(F, D, B, m_sol_red));
    triangles[curr_tris] = Triangle(F, D, B, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(F, H, D, m_sol_red));
    triangles[curr_tris] = Triangle(F, H, D, m_sol_red);
    curr_tris++;

    // BACK
    //triangles.push_back(Triangle(H, C, D, m_sol_red));
    triangles[curr_tris] = Triangle(H, C, D, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(H, G, C, m_sol_red));
    triangles[curr_tris] = Triangle(H, G, C, m_sol_red);
    curr_tris++;

    // LEFT
    //triangles.push_back(Triangle(G, E, C, m_sol_red));
    triangles[curr_tris] = Triangle(G, E, C, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(E, A, C, m_sol_red));
    triangles[curr_tris] = Triangle(E, A, C, m_sol_red);
    curr_tris++;

    // TOP
    //triangles.push_back(Triangle(G, F, E, m_sol_red));
    triangles[curr_tris] = Triangle(G, F, E, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(G, H, F, m_sol_red));
    triangles[curr_tris] = Triangle(G, H, F, m_sol_red);
    curr_tris++;

    // ---------------------------------------------------------------------------
    // Tall block

    A = vec4(443,0,247,1);
    B = vec4(285,0,296,1);
    C = vec4(492,0,406,1);
    D = vec4(334,0,456,1);

    E = vec4(443,330,247,1);
    F = vec4(285,330,296,1);
    G = vec4(492,330,406,1);
    H = vec4(334,330,456,1);

    // Front
   
    //triangles.push_back(Triangle(E, B, A, m_sol_blue));
    triangles[curr_tris] = Triangle(E, B, A, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(E, F, B, m_sol_blue));
    triangles[curr_tris] = Triangle(E, F, B, m_sol_blue);
    curr_tris++;

    // Front
    //triangles.push_back(Triangle(F, D, B, m_sol_blue));
    triangles[curr_tris] = Triangle(F, D, B, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(F, H, D, m_sol_blue));
    triangles[curr_tris] = Triangle(F, H, D, m_sol_blue);
    curr_tris++;

    // BACK
    //triangles.push_back(Triangle(H, C, D, m_sol_blue));
    triangles[curr_tris] = Triangle(H, C, D, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(H, G, C, m_sol_blue));
    triangles[curr_tris] = Triangle(H, G, C, m_sol_blue);
    curr_tris++;

    // LEFT
    //triangles.push_back(Triangle(G, E, C, m_sol_blue));
    triangles[curr_tris] = Triangle(G, E, C, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(E, A, C, m_sol_blue));
    triangles[curr_tris] = Triangle(E, A, C, m_sol_blue);
    curr_tris++;

    // TOP
    //triangles.push_back(Triangle(G, F, E, m_sol_blue));
    triangles[curr_tris] = Triangle(G, F, E, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(G, H, F, m_sol_blue));
    triangles[curr_tris] = Triangle(G, H, F, m_sol_blue);
    curr_tris++;

    // ---------------------------------------------------------------------------
    // Sphere

    //Sphere for the light
    spheres[0] = Sphere(vec4(0, -1.7, 0, 1), 1, m_light);

    // ----------------------------------------------
    // Scale to the volume [-1,1]^3

    for (size_t i = 0 ; i < curr_tris ; ++i) {
        triangles[i].v0_ = (triangles[i].v0_ * (2 / cornell_length));
        triangles[i].v1_ = (triangles[i].v1_ * (2 / cornell_length));
        triangles[i].v2_ = (triangles[i].v2_ * (2 / cornell_length));

        triangles[i].v0_ = (triangles[i].v0_ - vec4(1, 1, 1, 1));
        triangles[i].v1_ = (triangles[i].v1_ - vec4(1, 1, 1, 1));
        triangles[i].v2_ = (triangles[i].v2_ - vec4(1, 1, 1, 1));

        vec4 new_v0 = triangles[i].v0_;
        new_v0.x *= -1;
        new_v0.y *= -1;
        new_v0.w = 1.0;
        triangles[i].v0_ = (new_v0);

        vec4 new_v1 = triangles[i].v1_;
        new_v1.x *= -1;
        new_v1.y *= -1;
        new_v1.w = 1.0;
        triangles[i].v1_ = (new_v1);

        vec4 new_v2 = triangles[i].v2_;
        new_v2.x *= -1;
        new_v2.y *= -1;
        new_v2.w = 1.0;
        triangles[i].v2_ = (new_v2);

        triangles[i].computeAndSetNormal();
    }
}
