#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <SDL2/SDL.h>
#include <omp.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <chrono>

#include "constants/config.h"
#include "constants/materials.h"
#include "mcr.h"
#include "triangle.h"
#include "sphere.h"
#include "util.h"

int main (int argc, char* argv[]) {
    
    // Calculate the dimensions of the supersampled image
    const int supersample_width = screen_width * anti_aliasing_factor;
    const int supersample_height = screen_height * anti_aliasing_factor;

    // Pointer to the image on the host (CPU)
    vec3 * host_output = new vec3[supersample_width * supersample_height];

    // Pointer to the image on the device (GPU)
    vec3 * device_output;

    // Pointer to the aliased image on the host (CPU)
    vec3 * host_aliased_output = new vec3[screen_width * screen_height];

    // Pointer to the aliased image on the device (GPU)
    vec3 * device_aliased_output;

    // Allocate memory on CUDA device
    hipMalloc(&device_output, supersample_width * supersample_height * sizeof(vec3));
    hipMalloc(&device_aliased_output, screen_width * screen_height * sizeof(vec3));

    // Specify the block and grid dimensions to schedule CUDA threads
    dim3 threads_per_block(8, 8);
    dim3 num_blocks(
        supersample_width / threads_per_block.x,
        supersample_height / threads_per_block.y
    );

    // Create a vector of random states for use on the device
    hiprandState * device_rand_state;
    hipMalloc(
        (void **)&device_rand_state,
        supersample_width * supersample_height * sizeof(hiprandState)
    );

    // Load in the shapes
    int num_tris = 30;
    Triangle * triangles;
    hipMallocManaged(&triangles, num_tris * sizeof(Triangle));

    int num_spheres = 1;
    Sphere * spheres;
    hipMallocManaged(&spheres, num_tris * sizeof(Sphere));

    printf("CUDA has been initialised. Begin rendering...\n");
    printf("=============================================\n\n");

    // Load the polygons into the triangles array
    loadShapes(triangles, spheres);

    // Initialise the camera object
    Camera camera(
        cam_start_position,
        cam_start_yaw,
        cam_focal_length
    );

    // Define our area light
    LightSphere light_sphere(
        light_start_position, 
        area_light_radius, 
        num_lights, 
        light_intensity, 
        light_colour
    );

    auto start = std::chrono::high_resolution_clock::now();

    // Launch the CUDA kernel from the host and begin rendering 
    render_init<<<num_blocks, threads_per_block>>>(
        device_rand_state,
        supersample_height,
        supersample_width
    );

    render_kernel<<<num_blocks, threads_per_block>>>(
        device_output,
        supersample_height,
        supersample_width,
        camera,
        light_sphere,
        triangles,
        num_tris,
        spheres,
        num_spheres,
        device_rand_state
    ); 

    // Copy results of rendering back to the host
    hipMemcpy(
        host_output, 
        device_output, 
        supersample_width * supersample_height * sizeof(vec3), 
        hipMemcpyDeviceToHost
    ); 

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = end - start;
    int duration_in_ms = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

    printf("Finished rendering in %dms.\n", duration_in_ms);

    save_image(
        host_output, 
        supersample_height, 
        supersample_width, 
        pre_alias_title
    );

    // Specify different scheduling, this time we assign a thread to each pixel
    // of the output image
    threads_per_block = dim3(8, 8);
    num_blocks = dim3(
        screen_width / threads_per_block.x,
        screen_height / threads_per_block.y
    );

    // Perform anti aliasing
    MSAA<<<num_blocks, threads_per_block>>>(
        device_output,
        device_aliased_output,
        supersample_height,
        supersample_width
    );

    // Copy results of rendering back to the host
    hipMemcpy(
        host_aliased_output, 
        device_aliased_output, 
        screen_width * screen_height * sizeof(vec3), 
        hipMemcpyDeviceToHost
    ); 

    // Free CUDA memory
    hipFree(device_output); 
    hipFree(device_aliased_output); 
    
    printf("Finished aliasing!\n");

    // Save the aliased image
    save_image(
        host_aliased_output, 
        screen_height, 
        screen_width, 
        aliased_title
    );

    // Clear memory for host
    delete[] host_output;
    delete[] host_aliased_output;

    return 0;
}

// Initialises the random states for each thread with the same seed
__global__ 
void render_init(
    hiprandState * rand_state,
    int supersample_width,
    int supersample_height
) {
    // Assign a thread to each pixel (x, y)
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the pixel index in the linearised array
    unsigned int pixel_index = (supersample_height - y - 1) * supersample_width + x;

    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// Bulk of the rendering is controlled here
__global__
void render_kernel(
    vec3 * output,
    int supersample_width,
    int supersample_height,
    Camera camera,
    LightSphere light_sphere,
    Triangle * triangles,
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    hiprandState * rand_state
) {
    // Assign a cuda thread to each pixel (x,y)
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // The index of the pixel we are working on when the 2x2 array is linearised
    unsigned int pixel_index = (supersample_height - y - 1) * supersample_width + x;

    // Get the rand state for this thread
    hiprandState local_rand_state = rand_state[pixel_index];
    
    // Flip the y coordinate
    y = supersample_height - y;

    // Change the ray's direction to work for the current pixel (pixel space -> Camera space)
    vec4 dir(
        (float)x - supersample_width / 2 , 
        (float)y - supersample_height / 2 , 
        camera.focal_length_,
        1
    ); 

    // Create a ray for the given pixel
    Ray ray(camera.position_, dir);
    ray.rotateRay(camera.yaw_);

    // If the ray intersects with an object in the scene, perform monte carlo to
    // obtain a lighting estimate
    if (ray.closestIntersection(triangles, num_tris, spheres, num_spheres)) {

        vec3 colour = tracePath(
            ray.closest_intersection_,
            triangles,
            num_tris,
            spheres,
            num_spheres,
            local_rand_state,
            monte_carlo_max_depth,
            0
        );

        /*
        vec3 colour = monteCarlo(
            ray.closest_intersection_,
            triangles,
            num_tris,
            spheres,
            num_spheres,
            light_sphere,
            local_rand_state,
            monte_carlo_max_depth,
            0
        );
        */

        output[pixel_index] = colour;
    } 
    // if there is no intersection, we set the colour to be black
    else {
        output[pixel_index] = vec3(0.0f);
    }
}

__device__
vec3 tracePath(
    Intersection closest_intersection,
    Triangle * triangles,
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    hiprandState rand_state,
    int max_depth,
    int depth
) {
    if (depth >= max_depth) {
        return vec3(0.0f);
    } else {
        vec3 base_colour;
        // We have hit a triangle (not a light source)
        if (closest_intersection.is_triangle) {
            base_colour = triangles[closest_intersection.index].material_.diffuse_light_component_;

            vec3 intersection_normal_3 = vec3(closest_intersection.normal);
            vec3 N_t, N_b;
            createCoordinateSystem(intersection_normal_3, N_t, N_b);

            vec3 indirect_estimate = vec3(0);
            float pdf = 1 / (2 * (float)M_PI);
            for (int i = 0 ; i < monte_carlo_num_samples ; i++) {
                float r1 = hiprand_uniform(&rand_state); // cos(theta) = N.Light Direction
                float r2 = hiprand_uniform(&rand_state);
                vec3 sample = uniformSampleHemisphere(r1, r2);

                // Convert the sample from our coordinate space to world space
                vec4 sample_world(
                    sample.x * N_b.x + sample.y * intersection_normal_3.x + sample.z * N_t.x,
                    sample.x * N_b.y + sample.y * intersection_normal_3.y + sample.z * N_t.y,
                    sample.x * N_b.z + sample.y * intersection_normal_3.z + sample.z * N_t.z,
                    0
                );

                // Generate our ray from the random direction calculated previously
                Ray random_ray(
                    closest_intersection.position + sample_world * 0.0001f,
                    sample_world
                );

                if (random_ray.closestIntersection(triangles, num_tris, spheres, num_spheres)) {
                    indirect_estimate += r1 * tracePath(
                        random_ray.closest_intersection_,
                        triangles,
                        num_tris,
                        spheres,
                        num_spheres,
                        rand_state,
                        max_depth,
                        depth + 1
                    );
                }
            }
            indirect_estimate /= monte_carlo_num_samples * pdf;
            indirect_estimate *= base_colour;
            return indirect_estimate;
        }
        // We have hit a light source
        else {
            return vec3(4.0f);
            //base_colour = spheres[closest_intersection.index].material_.diffuse_light_component_;
        }

    }
}

// Calculates the indirect and direct light estimation for diffuse objects
__device__
vec3 monteCarlo(
    Intersection closest_intersection, 
    Triangle * triangles, 
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    LightSphere light_sphere,
    hiprandState rand_state,
    int max_depth,
    int depth
) {
    // If we have exceeded our limit of recursion, return the direct light at
    // this point multiplied by the object's colour
    if (depth >= max_depth) {
        vec3 direct_light = light_sphere.directLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres
        );

        vec3 base_colour;
        if (closest_intersection.is_triangle) {
            base_colour = triangles[closest_intersection.index].material_.diffuse_light_component_;
        } else {
            base_colour = spheres[closest_intersection.index].material_.diffuse_light_component_;
        }
        return direct_light * base_colour;
    } 
    // Otherwise, we must obtain an indirect lighting estimate for this point
    else {
        vec3 base_colour;
        if (closest_intersection.is_triangle) {
            base_colour = triangles[closest_intersection.index].material_.diffuse_light_component_;
        } else {
            base_colour = spheres[closest_intersection.index].material_.diffuse_light_component_;
        }
        vec3 direct_light = light_sphere.directLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres
        );
        vec3 indirect_estimate = indirectLight(
            closest_intersection,
            triangles,
            num_tris,
            spheres,
            num_spheres,
            light_sphere,
            rand_state,
            max_depth,
            depth + 1
        );
        return (direct_light + indirect_estimate) * base_colour;
    }
}

__device__
vec3 indirectLight(
    Intersection closest_intersection, 
    Triangle * triangles, 
    int num_tris,
    Sphere * spheres,
    int num_spheres,
    LightSphere light_sphere,
    hiprandState rand_state,
    int max_depth,
    int depth
) {
    vec3 intersection_normal_3 = vec3(closest_intersection.normal);
    
    vec3 N_t, N_b;
    createCoordinateSystem(intersection_normal_3, N_t, N_b);

    vec3 indirect_estimate = vec3(0);
    float pdf = 1 / (2 * (float)M_PI);
    for (int i = 0 ; i < monte_carlo_num_samples ; i++) {
        float r1 = hiprand_uniform(&rand_state); // cos(theta) = N.Light Direction
        float r2 = hiprand_uniform(&rand_state);
        vec3 sample = uniformSampleHemisphere(r1, r2);

        // Convert the sample from our coordinate space to world space
        vec4 sample_world(
            sample.x * N_b.x + sample.y * intersection_normal_3.x + sample.z * N_t.x,
            sample.x * N_b.y + sample.y * intersection_normal_3.y + sample.z * N_t.y,
            sample.x * N_b.z + sample.y * intersection_normal_3.z + sample.z * N_t.z,
            0
        );

        // Generate our ray from the random direction calculated previously
        Ray random_ray(
            closest_intersection.position + sample_world * 0.0001f,
            sample_world
        );

        if (random_ray.closestIntersection(triangles, num_tris, spheres, num_spheres)) {
               indirect_estimate += r1 * monteCarlo(
               random_ray.closest_intersection_,
               triangles,
               num_tris,
               spheres,
               num_spheres,
               light_sphere,
               rand_state,
               max_depth,
               depth + 1
            );
        }
    } 
    indirect_estimate /= monte_carlo_num_samples * pdf;
    return indirect_estimate;
}

// Given two random numbers between 0 and 1, return a direction to a point on a
// hemisphere
__device__
vec3 uniformSampleHemisphere(const float & r1, const float & r2) {
    // cos(theta) = r1 = y
    // cos^2(theta) + sin^2(theta) = 1 -> sin(theta) = srtf(1 - cos^2(theta))
    float sin_theta = sqrtf(1 - r1 * r1);
    float phi = 2 * (float)M_PI * r2;
    float x = sin_theta * cosf(phi);
    float z = sin_theta * sinf(phi);
    return vec3(x, r1, z);
} 

// This function creates a new coordinate system in which the up vector is
// oriented along the shaded point normal
__device__
void createCoordinateSystem(const vec3 & N, vec3 & N_t, vec3 & N_b) {
    if (std::fabs(N.x) > std::fabs(N.y)) {
        N_t = vec3(N.z, 0, -N.x) / sqrtf(N.x * N.x + N.z * N.z);
    } else {
        N_t = vec3(0, -N.z, N.y) / sqrtf(N.y * N.y + N.z * N.z);
    }
    N_b = glm::cross(N, N_t);
} 

__global__
void MSAA(
    vec3 * supersampled_image,
    vec3 * aliased_output,
    int supersample_height,
    int supersample_width
) {
    // Assign a cuda thread to each pixel (x,y)
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // The index of the pixel we are working on when the 2x2 array is linearised
    unsigned int output_pixel_index = (screen_height - y - 1) * screen_width + x;

    // Multiply x and y by the factor so that every pixel is included
    y *= anti_aliasing_factor;
    x *= anti_aliasing_factor;

    // Average the pixel values in a (factor^2 by factor^2) grid
    vec3 avg_pixel_value(0.0f);
    for (int i = 0 ; i < anti_aliasing_factor ; i++) {
        for (int j = 0 ; j < anti_aliasing_factor ; j++) {
            unsigned int input_pixel_index = (supersample_height - (y + i) - 1) * supersample_width + (x + j);
            avg_pixel_value += supersampled_image[input_pixel_index] / (float)(anti_aliasing_factor * anti_aliasing_factor);
        }
    }
    aliased_output[output_pixel_index] = avg_pixel_value;
}

void update(Camera & camera, Light & light) {
    static int t = SDL_GetTicks();
    /* Compute frame time */
    int t2 = SDL_GetTicks();
    float dt = float(t2-t);
    t = t2;

    std::cout << "Render time: " << dt << "ms." << std::endl;

    /* Update variables*/

    const Uint8* keystate = SDL_GetKeyboardState(NULL);

    if (keystate[SDL_SCANCODE_UP]) {
        camera.moveForwards(0.1);
    }
    if (keystate[SDL_SCANCODE_DOWN]) {
        camera.moveBackwards(0.1);
    }
    if (keystate[SDL_SCANCODE_LEFT]) {
        camera.rotateLeft(0.1);
    }
    if (keystate[SDL_SCANCODE_RIGHT]) {
        camera.rotateRight(0.1);
    }
    /*if (keystate[SDL_SCANCODE_A]) {
    }
    if (keystate[SDL_SCANCODE_D]) {
        light.translateRight(0.1);
    }
    if (keystate[SDL_SCANCODE_Q]) {
        light.translateUp(0.1);
    }
    if (keystate[SDL_SCANCODE_E]) {
        light.translateDown(0.1);
    }
    if (keystate[SDL_SCANCODE_W]) {
        light.translateForwards(0.1);
    }
    if (keystate[SDL_SCANCODE_S]) {
        light.translateBackwards(0.1);
    }*/
}

void loadShapes(Triangle * triangles, Sphere * spheres) {
    float cornell_length = 555;			// Length of Cornell Box side.

    vec4 A(cornell_length, 0, 0             , 1);
    vec4 B(0             , 0, 0             , 1);
    vec4 C(cornell_length, 0, cornell_length, 1);
    vec4 D(0             , 0, cornell_length, 1);

    vec4 E(cornell_length, cornell_length, 0             , 1);
    vec4 F(0             , cornell_length, 0             , 1);
    vec4 G(cornell_length, cornell_length, cornell_length, 1);
    vec4 H(0             , cornell_length, cornell_length, 1);

    // Counter to track triangles
    int curr_tris = 0;

    // Triangles now take a material as an argument rather than a colour
    // Floor:
    Triangle floor_triangle_1 = Triangle(C, B, A, m_sol_base3);
    //triangles.push_back(floor_triangle_1);
    triangles[curr_tris] = floor_triangle_1;
    curr_tris++;

    Triangle floor_triangle_2 = Triangle(C, D, B, m_sol_base3);
    //triangles.push_back(floor_triangle_2);
    triangles[curr_tris] = floor_triangle_2;
    curr_tris++;

    // Left wall
    Triangle left_wall_1 = Triangle(A, E, C, m_sol_base02);
    //triangles.push_back(left_wall_1);
    triangles[curr_tris] = left_wall_1;
    curr_tris++;

    Triangle left_wall_2 = Triangle(C, E, G, m_sol_base02);
    //triangles.push_back(left_wall_2);
    triangles[curr_tris] = left_wall_2;
    curr_tris++;

    // Right wall
    Triangle right_wall_1 = Triangle(F, B, D, m_sol_base02);
    //triangles.push_back(right_wall_1);
    triangles[curr_tris] = right_wall_1;
    curr_tris++;

    Triangle right_wall_2 = Triangle(H, F, D, m_sol_base02);
    //triangles.push_back(right_wall_2);
    triangles[curr_tris] = right_wall_2;
    curr_tris++;

    // Ceiling
    Triangle ceiling_1 = Triangle(E, F, G, m_sol_base01);
    //triangles.push_back(ceiling_1);
    triangles[curr_tris] = ceiling_1;
    curr_tris++;

    Triangle ceiling_2 = Triangle(F, H, G, m_sol_base01);
    //triangles.push_back(ceiling_2);
    triangles[curr_tris] = ceiling_2;
    curr_tris++;

    // Back wall
    Triangle back_wall_1 = Triangle(G, D, C, m_sol_yellow);
    //triangles.push_back(back_wall_1);
    triangles[curr_tris] = back_wall_1;
    curr_tris++;

    Triangle back_wall_2 = Triangle(G, H, D, m_sol_yellow);
    //triangles.push_back(back_wall_2);
    triangles[curr_tris] = back_wall_2;
    curr_tris++;

    // ---------------------------------------------------------------------------
    // Short block

    A = vec4(240,0,234,1);  //+120 in z -50 in x
    B = vec4( 80,0,185,1);
    C = vec4(190,0,392,1);
    D = vec4( 32,0,345,1);

    E = vec4(240,165,234,1);
    F = vec4( 80,165,185,1);
    G = vec4(190,165,392,1);
    H = vec4( 32,165,345,1);

    // Front
    //triangles.push_back(Triangle(E, B, A, m_sol_red));
    triangles[curr_tris] = Triangle(E, B, A, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(E, F, B, m_sol_red));
    triangles[curr_tris] = Triangle(E, F, B, m_sol_red);
    curr_tris++;

    // Front
    //triangles.push_back(Triangle(F, D, B, m_sol_red));
    triangles[curr_tris] = Triangle(F, D, B, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(F, H, D, m_sol_red));
    triangles[curr_tris] = Triangle(F, H, D, m_sol_red);
    curr_tris++;

    // BACK
    //triangles.push_back(Triangle(H, C, D, m_sol_red));
    triangles[curr_tris] = Triangle(H, C, D, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(H, G, C, m_sol_red));
    triangles[curr_tris] = Triangle(H, G, C, m_sol_red);
    curr_tris++;

    // LEFT
    //triangles.push_back(Triangle(G, E, C, m_sol_red));
    triangles[curr_tris] = Triangle(G, E, C, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(E, A, C, m_sol_red));
    triangles[curr_tris] = Triangle(E, A, C, m_sol_red);
    curr_tris++;

    // TOP
    //triangles.push_back(Triangle(G, F, E, m_sol_red));
    triangles[curr_tris] = Triangle(G, F, E, m_sol_red);
    curr_tris++;
    //triangles.push_back(Triangle(G, H, F, m_sol_red));
    triangles[curr_tris] = Triangle(G, H, F, m_sol_red);
    curr_tris++;

    // ---------------------------------------------------------------------------
    // Tall block

    A = vec4(443,0,247,1);
    B = vec4(285,0,296,1);
    C = vec4(492,0,406,1);
    D = vec4(334,0,456,1);

    E = vec4(443,330,247,1);
    F = vec4(285,330,296,1);
    G = vec4(492,330,406,1);
    H = vec4(334,330,456,1);

    // Front
   
    //triangles.push_back(Triangle(E, B, A, m_sol_blue));
    triangles[curr_tris] = Triangle(E, B, A, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(E, F, B, m_sol_blue));
    triangles[curr_tris] = Triangle(E, F, B, m_sol_blue);
    curr_tris++;

    // Front
    //triangles.push_back(Triangle(F, D, B, m_sol_blue));
    triangles[curr_tris] = Triangle(F, D, B, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(F, H, D, m_sol_blue));
    triangles[curr_tris] = Triangle(F, H, D, m_sol_blue);
    curr_tris++;

    // BACK
    //triangles.push_back(Triangle(H, C, D, m_sol_blue));
    triangles[curr_tris] = Triangle(H, C, D, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(H, G, C, m_sol_blue));
    triangles[curr_tris] = Triangle(H, G, C, m_sol_blue);
    curr_tris++;

    // LEFT
    //triangles.push_back(Triangle(G, E, C, m_sol_blue));
    triangles[curr_tris] = Triangle(G, E, C, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(E, A, C, m_sol_blue));
    triangles[curr_tris] = Triangle(E, A, C, m_sol_blue);
    curr_tris++;

    // TOP
    //triangles.push_back(Triangle(G, F, E, m_sol_blue));
    triangles[curr_tris] = Triangle(G, F, E, m_sol_blue);
    curr_tris++;
    //triangles.push_back(Triangle(G, H, F, m_sol_blue));
    triangles[curr_tris] = Triangle(G, H, F, m_sol_blue);
    curr_tris++;

    // ---------------------------------------------------------------------------
    // Sphere

    //Sphere for the right wall
    spheres[0] = Sphere(vec4(0, -1, -0.8, 1), 0.3, m_sol_green);

    // ----------------------------------------------
    // Scale to the volume [-1,1]^3

    for (size_t i = 0 ; i < curr_tris ; ++i) {
        triangles[i].v0_ = (triangles[i].v0_ * (2 / cornell_length));
        triangles[i].v1_ = (triangles[i].v1_ * (2 / cornell_length));
        triangles[i].v2_ = (triangles[i].v2_ * (2 / cornell_length));

        triangles[i].v0_ = (triangles[i].v0_ - vec4(1, 1, 1, 1));
        triangles[i].v1_ = (triangles[i].v1_ - vec4(1, 1, 1, 1));
        triangles[i].v2_ = (triangles[i].v2_ - vec4(1, 1, 1, 1));

        vec4 new_v0 = triangles[i].v0_;
        new_v0.x *= -1;
        new_v0.y *= -1;
        new_v0.w = 1.0;
        triangles[i].v0_ = (new_v0);

        vec4 new_v1 = triangles[i].v1_;
        new_v1.x *= -1;
        new_v1.y *= -1;
        new_v1.w = 1.0;
        triangles[i].v1_ = (new_v1);

        vec4 new_v2 = triangles[i].v2_;
        new_v2.x *= -1;
        new_v2.y *= -1;
        new_v2.w = 1.0;
        triangles[i].v2_ = (new_v2);

        triangles[i].computeAndSetNormal();
    }
}
